#include "hip/hip_runtime.h"
#include <stdio.h>   
#include "time.h"
#include "omp.h"

#include "emmintrin.h"
#include "nmmintrin.h"

#include "defs.h"
#include "func.h"

__constant__ int filter_laplace[5][5] = {-1, -1, -1, -1, -1,
                                         -1, -1, -1, -1, -1,
                                         -1, -1, 24, -1, -1,
                                         -1, -1, -1, -1, -1,
                                         -1, -1, -1, -1, -1};

__constant__ float filter_laplace_f[5][5] = {-1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, 24.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
                                             -1.0f, -1.0f, -1.0f, -1.0f, -1.0f};


// Glob�lis mem�ri�t haszn�l� (trivi�lis :)) megold�s
__global__ void kernel_conv_global(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
  
  // konvol�ci� 3 komponensre
  int acc_r = 0, acc_g = 0, acc_b = 0;

  for (int a = 0; a < FILTER_H; a++){
	  for (int b = 0; b < FILTER_W; b++) {
		  acc_r += gInput[((row + a) *imgWidthF + col + b) * 3] * filter_laplace[a][b];
		  acc_g += gInput[((row + a) *imgWidthF + col + b) * 3 + 1] * filter_laplace[a][b];
		  acc_b += gInput[((row + a) *imgWidthF + col + b) * 3 + 2] * filter_laplace[a][b];
	  }
  }

  // kimenet �r�sa
  if (acc_r > 255) gOutput[(row*imgWidth + col) * 3] = 255;
  else if (acc_r < 0) gOutput[(row*imgWidth + col) * 3] = 0;
  else gOutput[(row*imgWidth + col) * 3] = acc_r;

  if (acc_g > 255) gOutput[(row*imgWidth + col) * 3 + 1] = 255;
  else if (acc_g < 0) gOutput[(row*imgWidth + col) * 3 + 1] = 0;
  else gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;

  if (acc_b > 255) gOutput[(row*imgWidth + col) * 3 + 2] = 255;
  else if (acc_b < 0) gOutput[(row*imgWidth + col) * 3 + 2] = 0;
  else gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;

}

// Shared mem�ri�t haszn�l� megold�s (1.)
// Shared mem�ria adatt�pus: unsigned char
// Sz�m�t�s adatt�pusa: integer
__global__ void kernel_conv_sh_uchar_int(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + blockIdx.x * blockDim.x) * 3;


	// Sharde Memory deklar�ci�

	__shared__ unsigned char mem[20 * 20 * 3];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 240)
		for (int d = 0; d < 5; d++) mem[th1D + 240 * d] = gInput[base + th1D%60 + (4 * d + th1D / 60) * imgWidthF * 3];



	// Sz�l szinkroniz�ci�
	__syncthreads();


	// konvol�ci� 3 komponensre
	int acc_r = 0, acc_g = 0, acc_b = 0;

	for (int a = 0; a < FILTER_H; a++){
		for (int b = 0; b < FILTER_W; b++) {
			acc_r += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3] * filter_laplace[a][b];
			acc_g += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3 + 1] * filter_laplace[a][b];
			acc_b += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3 + 2] * filter_laplace[a][b];
		}
	}

	// kimenet �r�sa k�zvetlen�l a glob�lis mem�ri�ba
	if (acc_r > 255)
		gOutput[(row*imgWidth + col) * 3] = 255;
	else if (acc_r < 0)
		gOutput[(row*imgWidth + col) * 3] = 0;
	else
		gOutput[(row*imgWidth + col) * 3] = acc_r;

	if (acc_g > 255)
		gOutput[(row*imgWidth + col) * 3 + 1] = 255;
	else if (acc_g < 0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;

	if (acc_b > 255)
		gOutput[(row*imgWidth + col) * 3 + 2] = 255;
	else if (acc_b < 0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;

}

// Shared mem�ri�t haszn�l� megold�s (2.)
// Shared mem�ria adatt�pus: unsigned char
// Sz�m�t�s adatt�pusa: float
// A t�lt�s t�mb indexel�s helyett pointer + offset megold�ssal
__global__ void kernel_conv_sh_uchar_float(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + blockIdx.x * blockDim.x) * 3;


	// Sharde Memory deklar�ci�

	__shared__ unsigned char mem[20 * 20 * 3];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 240)
		for (int d = 0; d < 5; d++) mem[th1D + 240 * d] = gInput[base + th1D % 60 + (4 * d + th1D / 60) * imgWidthF * 3];



	// Sz�l szinkroniz�ci�
	__syncthreads();


	// konvol�ci� 3 komponensre
	float acc_r = 0, acc_g = 0, acc_b = 0;

	for (int a = 0; a < FILTER_H; a++){
		for (int b = 0; b < FILTER_W; b++) {
			acc_r += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3] * filter_laplace[a][b];
			acc_g += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3 + 1] * filter_laplace[a][b];
			acc_b += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3 + 2] * filter_laplace[a][b];
		}
	}

	// kimenet �r�sa k�zvetlen�l a glob�lis mem�ri�ba
	if (acc_r > 255.0)
		gOutput[(row*imgWidth + col) * 3] = 255;
	else if (acc_r < 0.0)
		gOutput[(row*imgWidth + col) * 3] = 0;
	else
		gOutput[(row*imgWidth + col) * 3] = acc_r;

	if (acc_g > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 255;
	else if (acc_g < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;

	if (acc_b > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 255;
	else if (acc_b < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;
}


// Shared mem�ri�t haszn�l� megold�s (3.)
// Shared mem�ria adatt�pus: float
// Sz�m�t�s adatt�pusa: float
__global__ void kernel_conv_sh_float_float(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + blockIdx.x * blockDim.x) * 3;


	// Sharde Memory deklar�ci�

	__shared__ float mem[20 * 20 * 3];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 240)
		for (int d = 0; d < 5; d++) mem[th1D + 240 * d] = gInput[base + th1D % 60 + (4 * d + th1D / 60) * imgWidthF * 3];

	// Sz�l szinkroniz�ci�
	__syncthreads();


	// konvol�ci� 3 komponensre
	float acc_r = 0, acc_g = 0, acc_b = 0;

	for (int a = 0; a < FILTER_H; a++){
		for (int b = 0; b < FILTER_W; b++) {
			acc_r += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3] * filter_laplace[a][b];
			acc_g += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3 + 1] * filter_laplace[a][b];
			acc_b += mem[(20 * (threadIdx.y + a) + threadIdx.x + b) * 3 + 2] * filter_laplace[a][b];
		}
	}

	// kimenet �r�sa k�zvetlen�l a glob�lis mem�ri�ba
	if (acc_r > 255.0)
		gOutput[(row*imgWidth + col) * 3] = 255;
	else if (acc_r < 0.0)
		gOutput[(row*imgWidth + col) * 3] = 0;
	else
		gOutput[(row*imgWidth + col) * 3] = acc_r;

	if (acc_g > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 255;
	else if (acc_g < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;

	if (acc_b > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 255;
	else if (acc_b < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;

}



// Ugyanaz mint az el�bb, m�dos�tott blokk m�retekkel (16*16) az olvas�si shared memory bank konfliktus elker�l�s�hez
__global__ void kernel_conv_sh_float_float_nbc(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + blockIdx.x * blockDim.x) * 3;


	// Sharde Memory deklar�ci�

	__shared__ float mem[20 * 65];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 240)
		for (int d = 0; d < 5; d++) mem[th1D + 65 * 4 * d + (th1D / 60 * 5)] = gInput[base + th1D % 60 + (4 * d + th1D / 60) * imgWidthF * 3];

	// Sz�l szinkroniz�ci�
	__syncthreads();


	// konvol�ci� 3 komponensre
	float acc_r = 0, acc_g = 0, acc_b = 0;

	for (int a = 0; a < FILTER_H; a++){
		for (int b = 0; b < FILTER_W; b++) {
			acc_r += mem[65 * (threadIdx.y + a) + (threadIdx.x + b) * 3] * filter_laplace[a][b];
			acc_g += mem[65 * (threadIdx.y + a) + (threadIdx.x + b) * 3 + 1] * filter_laplace[a][b];
			acc_b += mem[65 * (threadIdx.y + a) + (threadIdx.x + b) * 3 + 2] * filter_laplace[a][b];
		}
	}

	/*acc_r = mem[65 * (threadIdx.y) + (threadIdx.x) * 3];
	acc_g = mem[65 * (threadIdx.y) + (threadIdx.x) * 3 + 1];
	acc_b = mem[65 * (threadIdx.y) + (threadIdx.x) * 3 + 2];
	*/
	// kimenet �r�sa k�zvetlen�l a glob�lis mem�ri�ba
	if (acc_r > 255.0)
		gOutput[(row*imgWidth + col) * 3] = 255;
	else if (acc_r < 0.0)
		gOutput[(row*imgWidth + col) * 3] = 0;
	else
		gOutput[(row*imgWidth + col) * 3] = acc_r;

	if (acc_g > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 255;
	else if (acc_g < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;

	if (acc_b > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 255;
	else if (acc_b < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;
}


// Ugyanaz mint az el�bb, m�dos�tott blokk m�retekkel (32x8) az olvas�si shared memory bank konfliktus elker�l�s�hez
__global__ void kernel_conv_sh_float_float_nbc_easy(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + blockIdx.x * blockDim.x) * 3;


	// Sharde Memory deklar�ci�

	__shared__ float mem[12][36][3];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 216)
		for (int d = 0; d < 6; d++)
			mem[th1D / (36 * 3) + d * 2][(th1D / 3) % 36][th1D % 3] = gInput[base + th1D % 108 + (th1D / (36 * 3) + d * 2) * imgWidthF * 3];

	// Sz�l szinkroniz�ci�
	__syncthreads();


	// konvol�ci� 3 komponensre
	float acc_r = 0, acc_g = 0, acc_b = 0;

	for (int a = 0; a < FILTER_H; a++){
		for (int b = 0; b < FILTER_W; b++) {
			acc_r += mem[threadIdx.y + a][threadIdx.x + b][0] * filter_laplace[a][b];
			acc_g += mem[threadIdx.y + a][threadIdx.x + b][1] * filter_laplace[a][b];
			acc_b += mem[threadIdx.y + a][threadIdx.x + b][2] * filter_laplace[a][b];
		}
	}

	/*acc_r = mem[65 * (threadIdx.y) + (threadIdx.x) * 3];
	acc_g = mem[65 * (threadIdx.y) + (threadIdx.x) * 3 + 1];
	acc_b = mem[65 * (threadIdx.y) + (threadIdx.x) * 3 + 2];
	*/
	// kimenet �r�sa k�zvetlen�l a glob�lis mem�ri�ba
	if (acc_r > 255.0)
		gOutput[(row*imgWidth + col) * 3] = 255;
	else if (acc_r < 0.0)
		gOutput[(row*imgWidth + col) * 3] = 0;
	else
		gOutput[(row*imgWidth + col) * 3] = acc_r;

	if (acc_g > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 255;
	else if (acc_g < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 1] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;

	if (acc_b > 255.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 255;
	else if (acc_b < 0.0)
		gOutput[(row*imgWidth + col) * 3 + 2] = 0;
	else
		gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;
}

// Ugyanaz mint az el�bb, m�dos�tott blokk m�retekkel (16*16) az olvas�si shared memory bank konfliktus elker�l�s�hez
__global__ void kernel_median(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = threadIdx.x + blockDim.x * blockIdx.x;   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + blockIdx.x * blockDim.x) * 3;


	// Sharde Memory deklar�ci�

	__shared__ float mem[20 * 65];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 240)
		for (int d = 0; d < 5; d++) mem[th1D + 65 * 4 * d + (th1D / 60 * 5)] = gInput[base + th1D % 60 + (4 * d + th1D / 60) * imgWidthF * 3];

	// Sz�l szinkroniz�ci�
	__syncthreads();


	// konvol�ci� 3 komponensre
	float acc_r = 0, acc_g = 0, acc_b = 0;

	for (int a = 0; a < FILTER_H; a++){
		for (int b = 0; b < FILTER_W; b++) {
			acc_r += mem[65 * (threadIdx.y + a) + (threadIdx.x + b) * 3] * filter_laplace[a][b];
			acc_g += mem[65 * (threadIdx.y + a) + (threadIdx.x + b) * 3 + 1] * filter_laplace[a][b];
			acc_b += mem[65 * (threadIdx.y + a) + (threadIdx.x + b) * 3 + 2] * filter_laplace[a][b];
		}
	}

	// kimenet �r�sa k�zvetlen�l a glob�lis mem�ri�ba

		gOutput[(row*imgWidth + col) * 3] = acc_r;
		gOutput[(row*imgWidth + col) * 3 + 1] = acc_g;
		gOutput[(row*imgWidth + col) * 3 + 2] = acc_b;
}


__global__ void kernel_median_char(unsigned char* gInput, unsigned char* gOutput, int imgWidth, int imgWidthF)
{
	register unsigned int tmp;
#define swap(a, b) {if(a>b) {tmp=a; a=b; b=tmp;}}

	register unsigned int sort[25];


	int row = threadIdx.y + blockDim.y * blockIdx.y;	// a sz�l melyik sorban lev� kimeneti pixelt sz�molja
	int col = 2 * (threadIdx.x + blockDim.x * blockIdx.x);   // a sz�l melyik oszlopban lev� kimeneti pixelt sz�molja
	int base = (blockIdx.y * blockDim.y * imgWidthF + 2 * blockIdx.x * blockDim.x) * 3;

	// Sharde Memory deklar�ci�
	__shared__ unsigned char mem[36 * 20 * 3];

	// Shared Memory felt�lt�s
	int th1D = blockDim.x * threadIdx.y + threadIdx.x;	// lie�ris sz�l-azonos�t� a Thread Block-on bel�l
	if (th1D < 216)
		for (int d = 0; d < 10; d++)
			mem[th1D + 216 * d] = gInput[base + th1D % 108 + (2 * d + th1D / 108) * imgWidthF * 3];

	// Sz�l szinkroniz�ci�
	__syncthreads();

	for (int i = 0; i < 3; i++){
		for (int a = 0; a < FILTER_H; a++)
			for (int b = 0; b < FILTER_W; b++)
				sort[a * 5 + b] = mem[(36 * (threadIdx.y + a) + 2 * threadIdx.x + b) * 3 + i];
		//1
		swap(sort[1], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]); swap(sort[11], sort[12]); swap(sort[13], sort[14]); swap(sort[16], sort[17]);
		swap(sort[1], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]); swap(sort[11], sort[13]); swap(sort[12], sort[14]);
		swap(sort[1], sort[6]); swap(sort[4], sort[9]); swap(sort[11], sort[16]); swap(sort[14], sort[17]);
		swap(sort[1], sort[11]); swap(sort[9], sort[17]);
		
		//2
		swap(sort[18], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]); swap(sort[11], sort[12]); swap(sort[13], sort[14]);
		swap(sort[18], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]); swap(sort[11], sort[13]); swap(sort[12], sort[14]);
		swap(sort[18], sort[6]); swap(sort[4], sort[9]); swap(sort[11], sort[16]); swap(sort[14], sort[16]);
		swap(sort[18], sort[11]); swap(sort[9], sort[16]);
		
		//3
		swap(sort[19], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]); swap(sort[11], sort[12]); swap(sort[13], sort[14]);
		swap(sort[19], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]); swap(sort[11], sort[13]); swap(sort[12], sort[14]);
		swap(sort[19], sort[6]); swap(sort[4], sort[9]); swap(sort[11], sort[14]);
		swap(sort[19], sort[11]); swap(sort[9], sort[14]);
		
		//4
		swap(sort[21], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]); swap(sort[11], sort[12]);
		swap(sort[21], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]); swap(sort[12], sort[13]);
		swap(sort[21], sort[6]); swap(sort[4], sort[9]); swap(sort[11], sort[13]);
		swap(sort[21], sort[11]); swap(sort[9], sort[13]);
		
		//5
		swap(sort[22], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]);
		swap(sort[22], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]);
		swap(sort[22], sort[6]); swap(sort[4], sort[9]); swap(sort[11], sort[12]);
		swap(sort[22], sort[11]); swap(sort[9], sort[12]);
		
		//6
		swap(sort[23], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]);
		swap(sort[23], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]);
		swap(sort[23], sort[6]); swap(sort[4], sort[9]);
		swap(sort[23], sort[11]); swap(sort[9], sort[11]);
		
		//7
		swap(sort[24], sort[2]); swap(sort[3], sort[4]); swap(sort[6], sort[7]); swap(sort[8], sort[9]);
		swap(sort[24], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[8]); swap(sort[7], sort[9]);
		swap(sort[24], sort[6]); swap(sort[4], sort[9]);
		swap(sort[24], sort[9]);
		
		//Counting second pixel, too
		//0,2,3,4,6,7,8,5,10,15,20
		sort[11] = sort[2];
		sort[12] = sort[3];
		sort[13] = sort[4];
		sort[14] = sort[6];
		sort[16] = sort[7];
		sort[17] = sort[8];
		sort[18] = mem[(36 * (threadIdx.y) + 2*threadIdx.x + FILTER_W) * 3 + i];
		sort[19] = mem[(36 * (threadIdx.y + 1) + 2*threadIdx.x + FILTER_W) * 3 + i];
		sort[21] = mem[(36 * (threadIdx.y + 2) + 2*threadIdx.x + FILTER_W) * 3 + i];
		sort[22] = mem[(36 * (threadIdx.y + 3) + 2*threadIdx.x + FILTER_W) * 3 + i];
		sort[23] = mem[(36 * (threadIdx.y + 4) + 2*threadIdx.x + FILTER_W) * 3 + i];
		
		//8
		swap(sort[0], sort[2]); swap(sort[3], sort[4]); swap(sort[14], sort[7]);
		swap(sort[0], sort[3]); swap(sort[2], sort[4]); swap(sort[14], sort[8]); swap(sort[7], sort[8]);
		swap(sort[0], sort[14]); swap(sort[4], sort[8]);
		swap(sort[0], sort[8]);
		//8-2
		swap(sort[18], sort[11]); swap(sort[12], sort[13]); swap(sort[6], sort[16]);
		swap(sort[18], sort[12]); swap(sort[11], sort[13]); swap(sort[6], sort[17]); swap(sort[16], sort[17]);
		swap(sort[18], sort[6]); swap(sort[13], sort[17]);
		swap(sort[18], sort[17]);
		
		//9
		swap(sort[5], sort[2]); swap(sort[3], sort[4]);
		swap(sort[5], sort[3]); swap(sort[2], sort[4]); swap(sort[6], sort[7]);
		swap(sort[5], sort[6]); swap(sort[4], sort[7]);
		swap(sort[5], sort[7]);
		//9-2
		swap(sort[19], sort[11]); swap(sort[12], sort[13]);
		swap(sort[19], sort[12]); swap(sort[11], sort[13]); swap(sort[14], sort[16]);
		swap(sort[19], sort[14]); swap(sort[13], sort[16]);
		swap(sort[19], sort[16]);
		
		//10
		swap(sort[10], sort[2]); swap(sort[3], sort[4]);
		swap(sort[10], sort[3]); swap(sort[2], sort[4]);
		swap(sort[4], sort[6]);
		swap(sort[10], sort[6]);
		//10-2
		swap(sort[21], sort[11]); swap(sort[12], sort[13]);
		swap(sort[21], sort[12]); swap(sort[11], sort[13]);
		swap(sort[13], sort[14]);
		swap(sort[21], sort[14]);
		
		//11
		swap(sort[15], sort[2]); swap(sort[3], sort[4]);
		swap(sort[15], sort[3]); swap(sort[2], sort[4]);
		swap(sort[15], sort[4]);
		//11-2
		swap(sort[22], sort[11]); swap(sort[12], sort[13]);
		swap(sort[22], sort[12]); swap(sort[11], sort[13]);
		swap(sort[22], sort[13]);
		
		//12
		swap(sort[20], sort[2]);
		swap(sort[2], sort[3]);
		swap(sort[20], sort[3]);
		//12-2
		swap(sort[23], sort[11]);
		swap(sort[11], sort[12]);
		swap(sort[23], sort[12]);

		gOutput[(row*imgWidth + col) * 3 + i] =  sort[2];
		gOutput[(row*imgWidth + col + 1) * 3 + i] = sort[11];
	}
}


void cudaMain(int imgHeight, int imgWidth, int imgHeightF, int imgWidthF,
			  int imgFOfssetH, int imgFOfssetW,
			  unsigned char *imgSrc, unsigned char *imgDst)

{
    double s0, e0;
    double d0;


    int size_in  = imgWidthF*imgHeightF*sizeof(unsigned char) * 3;
	int size_out = imgWidth*imgHeight*sizeof(unsigned char) * 3;

    unsigned char *gInput, *gOutput;
	hipMalloc((void**)&gInput, size_in);
	hipMalloc((void**)&gOutput, size_out);

	dim3 thrBlock(16, 16);
    dim3 thrGrid(imgWidth/16, imgHeight/16);

	dim3 thrBlock2(32, 8);
	dim3 thrGrid2(imgWidth / 32, imgHeight / 8);
	
	dim3 thrBlock3(16, 16);
	dim3 thrGrid3(imgWidth / 32, imgHeight / 16);

	hipMemcpy(gInput, imgSrc, size_in, hipMemcpyHostToDevice); 

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	s0 = time_measure(1);
	for (int i = 0; i < KERNEL_RUNS; i++)
	{
		//kernel_conv_global << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_uchar_int << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_uchar_float << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_float_float << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_float_float_nbc << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_conv_sh_float_float_nbc_easy << <thrGrid2, thrBlock2 >> >(gInput, gOutput, imgWidth, imgWidthF);
		//kernel_median << <thrGrid, thrBlock >> >(gInput, gOutput, imgWidth, imgWidthF);
		kernel_median_char << <thrGrid3, thrBlock3 >> >(gInput, gOutput, imgWidth, imgWidthF);
	}
	hipDeviceSynchronize();
	e0 = time_measure(2);

    hipMemcpy(imgDst, gOutput, size_out, hipMemcpyDeviceToHost);
	
    hipFree(gInput); hipFree(gOutput);

	hipDeviceReset();

    d0 = (double)(e0-s0)/(CLOCKS_PER_SEC*KERNEL_RUNS);
	double mpixel = (imgWidth*imgHeight / d0) / 1000000;
    printf("CUDA single kernel time: %4.4f\n", d0);
	printf("CUDA Mpixel/s: %4.4f\n", mpixel);
}